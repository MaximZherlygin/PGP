
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

using namespace std;

__global__ void kernel(double *res, double *arr1, double *arr2, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;			// Абсолютный номер потока
    int offset = blockDim.x * gridDim.x;						// Общее кол-во потоков
    for(int i = idx; i < n; i += offset) {
        res[i] = arr1[i] * arr2[i];
    }
}

int main() {
    int size;
    scanf("%d", &size);
    double *res = (double *)malloc(sizeof(double) * size);
    double *vec1 = (double *)malloc(sizeof(double) * size);
    double *vec2 = (double *)malloc(sizeof(double) * size);
    for(int i = 0; i < size; i++) {
        scanf("%lf", &vec1[i]);
    }
    for(int i = 0; i < size; i++) {
        scanf("%lf", &vec2[i]);
    }

    double *devRes, *devVec1, *devVec2;
    hipMalloc(&devRes, sizeof(double) * size);
    hipMemcpy(devRes, res, sizeof(double) * size, hipMemcpyHostToDevice);

    hipMalloc(&devVec1, sizeof(double) * size);
    hipMemcpy(devVec1, vec1, sizeof(double) * size, hipMemcpyHostToDevice);

    hipMalloc(&devVec2, sizeof(double) * size);
    hipMemcpy(devVec2, vec2, sizeof(double) * size, hipMemcpyHostToDevice);


    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    kernel<<<256, 256>>>(devRes, devVec1, devVec2, size);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    fprintf(stderr, "time = %f\n", time);
    hipEventDestroy(stop);
    hipEventDestroy(start);


    // cudaMemcpy(res, devRes, sizeof(double) * n, cudaMemcpyDeviceToHost);
    hipFree(devRes);
    hipFree(devVec1);
    hipFree(devVec2);
//    for(int i = 0; i < n; i++) {
//        printf("%f ", res[i]);
//    }
    printf("\n");
    free(res);
    free(vec1);
    free(vec2);
    return 0;
}
