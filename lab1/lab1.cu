
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

using namespace std;

__global__ void kernel(double *arr1, double *arr2, int size) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;			// Абсолютный номер потока
    int offset = blockDim.x * gridDim.x;						// Общее кол-во потоков
    for(int i = idx; i < size; i += offset) {
        arr1[i] = arr1[i] * arr2[i];
    }
}

int main() {
    int size;
    scanf("%d", &size);
    double *vec1 = (double *)malloc(sizeof(double) * size);
    double *vec2 = (double *)malloc(sizeof(double) * size);
    for(int i = 0; i < size; i++) {
        scanf("%lf", &vec1[i]);
    }
    for(int i = 0; i < size; i++) {
        scanf("%lf", &vec2[i]);
    }

    double *devVec1, *devVec2;

    hipMalloc(&devVec1, sizeof(double) * size);
    hipMemcpy(devVec1, vec1, sizeof(double) * size, hipMemcpyHostToDevice);

    hipMalloc(&devVec2, sizeof(double) * size);
    hipMemcpy(devVec2, vec2, sizeof(double) * size, hipMemcpyHostToDevice);


    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    kernel<<<256, 256>>>(devVec1, devVec2, size);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    fprintf(stderr, "time = %f\n", time);
    hipEventDestroy(stop);
    hipEventDestroy(start);


    hipMemcpy(vec1, devVec1, sizeof(double) * size, hipMemcpyDeviceToHost);
    for(int i = 0; i < size; i++) {
        printf("%f ", vec1[i]);
    }
    printf("\n");
    hipFree(devVec1);
    hipFree(devVec2);
    free(vec1);
    free(vec2);
    return 0;
}
