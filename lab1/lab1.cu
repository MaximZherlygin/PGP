
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>

using namespace std;

#define HANDLE_ERROR(err)                             \
    do {                                              \
        if (err != hipSuccess) {                     \
            printf("ERROR: %s\n", hipGetErrorString(err)); \
            exit(0);                                  \
        }                                             \
    } while (0)

__global__ void kernel(double *res, double *arr1, double *arr2, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;			// Абсолютный номер потока
    int offset = blockDim.x * gridDim.x;						// Общее кол-во потоков
    for(int i = idx; i < n; i += offset) {
        res[i] = arr1[i] * arr2[i];
    }
}

int main()
{
    std::ios_base::sync_with_stdio(false);

    int size = 0;
    std::cin >> size;

    double *vec1 = new double[size];
    double *vec2 = new double[size];
    double *res = new double[size];

    for (int i = 0; i < size; ++i) {
        std::cin >> vec1[i];
    }
    for (int i = 0; i < size; ++i) {
        std::cin >> vec2[i];
    }

    double *dev1, *dev2, *devRes;

    HANDLE_ERROR(hipMalloc((void **) &dev1, sizeof(double) * size));
    HANDLE_ERROR(hipMalloc((void **) &dev2, sizeof(double) * size));
    HANDLE_ERROR(hipMalloc((void **) &devRes, sizeof(double) * size));

    HANDLE_ERROR(hipMemcpy(dev1, vec1, sizeof(double) * size, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev2, vec2, sizeof(double) * size, hipMemcpyHostToDevice));

    kernel<<<256, 256>>>(dev1, dev2, devRes, size);
    HANDLE_ERROR(hipGetLastError());

    HANDLE_ERROR(hipMemcpy(res, devRes, sizeof(double) * size, hipMemcpyDeviceToHost));

    std::cout.precision(10);
    std::cout.setf(std::ios::scientific);
    for (int i = 0; i < size; ++i) {
        std::cout << res[i] << ' ';
    }
    std::cout << '\n';

    HANDLE_ERROR(hipFree(dev1));
    HANDLE_ERROR(hipFree(dev2));
    HANDLE_ERROR(hipFree(devRes));

    delete[] vec1;
    delete[] vec2;
    delete[] res;
}
