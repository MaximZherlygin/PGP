#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <float.h>
#include <vector>
#include <string>
#include <chrono>
#include "hip/hip_runtime.h"
#include ""

#define _USE_MATH_DEFINES // for C++
#include <cmath>

using namespace std;

#define CUDA_ERROR(err) { \
    if (err != hipSuccess) { \
        fprintf(stderr, "ERROR: CUDA failed in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        return(1); \
    } \
} \

typedef unsigned char uchar;

struct vec3 {
    double x;
    double y;
    double z;
};

struct polygon {
    vec3 p1;
    vec3 p2;
    vec3 p3;
    vec3 color;
};

__host__ __device__  vec3 operator + (vec3 v1, vec3 v2) {
    return  vec3{v1.x + v2.x,
                         v1.y + v2.y,
                         v1.z + v2.z};
}

__host__ __device__  vec3 operator - (vec3 v1, vec3 v2) {
    return  vec3{v1.x - v2.x,
                         v1.y - v2.y,
                         v1.z - v2.z};
}

__host__ __device__  vec3 operator * (vec3 v, double num) {
    return vec3{v.x * num,
                        v.y * num,
                        v.z * num};
}

__host__ __device__  double scal_mul(vec3 v1, vec3 v2) {
    return v1.x * v2.x + v1.y * v2.y + v1.z * v2.z;
}

__host__ __device__  double len(vec3 v) {
    return sqrt(scal_mul(v, v));
}

__host__ __device__  vec3 norm(vec3 v) {
    double num = len(v);
    return vec3{v.x / num,
                        v.y / num,
                        v.z / num};
}

__host__ __device__ vec3 crossing(vec3 v1, vec3 v2) {
    return {v1.y * v2.z - v1.z * v2.y,
            v1.z * v2.x - v1.x * v2.z,
            v1.x * v2.y - v1.y * v2.x};
}

__host__ __device__ vec3 multiply(vec3 a, vec3 b, vec3 c, vec3 v) {
    return { a.x * v.x + b.x * v.y + c.x * v.z,
             a.y * v.x + b.y * v.y + c.y * v.z,
             a.z * v.x + b.z * v.y + c.z * v.z };
}

vec3 normalise_color(vec3 color) {
    return {color.x * 255.,
            color.y * 255.,
            color.z * 255.};
}

__host__ __device__ uchar4 ray_aux(vec3 pos, vec3 dir, vec3 light_pos,
                                   vec3 light_color, polygon *polygons, int n) {
    int min_value = -1;
    double ts_min;
    for (int i = 0; i < n; ++i) {
        vec3 e1 = polygons[i].p2 - polygons[i].p1;
        vec3 e2 = polygons[i].p3 - polygons[i].p1;
        vec3 p = crossing(dir, e2);
        double dwiv = scal_mul(p, e1);

        if (fabs(div) < 1e-10)
            continue;

        vec3 t = pos - polygons[i].p1;
        double u = scal_mul(p, t) / div;
        if (u < 0.0 || u > 1.0)
            continue;

        vec3 q = crossing(t, e1);
        double v = scal_mul(q, dir) / div;
        if (v < 0.0 || v + u > 1.0)
            continue;

        double ts = scal_mul(q, e2) / div;
        if (ts < 0.0)
            continue;

        if (min_value == -1 || ts < ts_min) {
            min_value = i;
            ts_min = ts;
        }
    }

    if (min_value == -1)
        return {0, 0, 0, 0};

    pos = dir * ts_min + pos;
    dir = light_pos - pos;
    double length = len(dir);
    dir = norm(dir);

    for (int i = 0; i < n; i++) {
        vec3 e1 = polygons[i].p2 - polygons[i].p1;
        vec3 e2 = polygons[i].p3 - polygons[i].p1;
        vec3 p = crossing(dir, e2);
        double div = scal_mul(p, e1);

        if (fabs(div) < 1e-10)
            continue;

        vec3 t = pos - polygons[i].p1;
        double u = scal_mul(p, t) / div;

        if (u < 0.0 || u > 1.0)
            continue;

        vec3 q = crossing(t, e1);
        double v = scal_mul(q, dir) / div;

        if (v < 0.0 || v + u > 1.0)
            continue;

        double ts = scal_mul(q, e2) / div;

        if (ts > 0.0 && ts < length && i != min_value) {
            return {0, 0, 0, 0};
        }
    }

    uchar4 color_min;
    color_min.x = polygons[min_value].color.x;
    color_min.y = polygons[min_value].color.y;
    color_min.z = polygons[min_value].color.z;

    color_min.x *= light_color.x;
    color_min.y *= light_color.y;
    color_min.z *= light_color.z;
    color_min.w = 0;
    return color_min;
}

void render_cpu(vec3 p_c, vec3 p_v, int w, int h, double fov, uchar4* pixels, vec3 light_pos,
                vec3 light_col, polygon* polygons, int n) {
    double dw = (double)2.0 / (double)(w - 1.0);
    double dh = (double)2.0 / (double)(h - 1.0);
    double z = 1.0 / tan(fov * M_PI / 360.0);
    vec3 b_z = norm(p_v - p_c);
    vec3 b_x = norm(crossing(b_z, {0.0, 0.0, 1.0}));
    vec3 b_y = norm(crossing(b_x, b_z));
    for (int i = 0; i < w; i++)
        for (int j = 0; j < h; j++) {
            vec3 v;
            v.x = (double)-1.0 + dw * (double)i;
            v.y = ((double)-1.0 + dh * (double)j) * (double)h / (double)w;
            v.z = z;
            vec3 dir = multiply(b_x, b_y, b_z, v);
            pixels[(h - 1 - j) * w + i] = ray_aux(p_c, norm(dir), light_pos, light_col, polygons, n);
        }
}

__global__ void render_gpu(vec3 p_c, vec3 p_v, int w, int h, double fov, uchar4* pixels,
                           vec3 light_pos, vec3 light_col, polygon* polygons, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    int offsetX = blockDim.x * gridDim.x;
    int offsetY = blockDim.y * gridDim.y;

    double dw = (double)2.0 / (double)(w - 1.0);
    double dh = (double)2.0 / (double)(h - 1.0);
    double z = 1.0 / tan(fov * M_PI / 360.0);
    vec3 b_z = norm(p_v - p_c);
    vec3 b_x = norm(crossing(b_z, {0.0, 0.0, 1.0}));
    vec3 b_y = norm(crossing(b_x, b_z));
    for (int i = idx; i < w; i += offsetX)
        for (int j = idy; j < h; j += offsetY) {
            vec3 v;
            v.x = (double)-1.0 + dw * (double)i;
            v.y = ((double)-1.0 + dh * (double)j) * (double)h / (double)w;
            v.z = z;
            vec3 dir = multiply(b_x, b_y, b_z, v);
            pixels[(h - 1 - j) * w + i] = ray_aux(p_c, norm(dir), light_pos, light_col, polygons, n);
        }
}

void ssaa_cpu(uchar4 *pixels, int w, int h, int coeff, uchar4 *ssaa_pixels) {
    for (int y = 0; y < h; y++) {
        for (int x = 0; x < w; x++) {
            int4 mid_pixel = { 0, 0, 0, 0 };
            for (int j = 0; j < coeff; j++) {
                for (int i = 0; i < coeff; i++) {
                    int index = y * w * coeff * coeff + x * coeff + j * w * coeff + i;
                    mid_pixel.x += ssaa_pixels[index].x;
                    mid_pixel.y += ssaa_pixels[index].y;
                    mid_pixel.z += ssaa_pixels[index].z;
                    mid_pixel.w += 0;
                }
            }
            pixels[y * w + x].x = (uchar)(int)(mid_pixel.x / (coeff * coeff));
            pixels[y * w + x].y = (uchar)(int)(mid_pixel.y / (coeff * coeff));
            pixels[y * w + x].z = (uchar)(int)(mid_pixel.z / (coeff * coeff));
            pixels[y * w + x].w = mid_pixel.w;
        }
    }
}

__global__ void ssaa_gpu(uchar4 *pixels, int w, int h, int coeff, uchar4 *ssaa_pixels) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int offsetX = blockDim.x * gridDim.x;
    int offsetY = blockDim.y * gridDim.y;

    for (int y = idy; y < h; y += offsetY) {
        for (int x = idx; x < w; x += offsetX) {
            int4 mid = { 0, 0, 0, 0 };
            for (int j = 0; j < coeff; j++) {
                for (int i = 0; i < coeff; i++) {
                    int index = y * w * coeff * coeff + x * coeff + j * w * coeff + i;
                    mid.x += ssaa_pixels[index].x;
                    mid.y += ssaa_pixels[index].y;
                    mid.z += ssaa_pixels[index].z;
                    mid.w += 0;
                }
            }
            pixels[y * w + x].x = (uchar)(mid.x / (coeff * coeff));
            pixels[y * w + x].y = (uchar)(mid.y / (coeff * coeff));
            pixels[y * w + x].z = (uchar)(mid.z / (coeff * coeff));
            pixels[y * w + x].w = mid.w;
        }
    }
}

void hexahedron(vec3 center, double r, vec3 color, vector<polygon> &polygons) { // ++++++
    // cout << "Creating hexahedron\n";

    color = normalise_color(color);

    // Create all vertices
    vector<vec3> vertices(8);

    vec3 point_a {-1 / sqrt(3), -1 / sqrt(3), -1 / sqrt(3)};
    vec3 point_b {-1 / sqrt(3), -1 / sqrt(3), 1 / sqrt(3)};
    vec3 point_c {-1 / sqrt(3), 1 / sqrt(3), -1 / sqrt(3)};
    vec3 point_d {-1 / sqrt(3), 1 / sqrt(3), 1 / sqrt(3)};
    vec3 point_e {1 / sqrt(3), -1 / sqrt(3), -1 / sqrt(3)};
    vec3 point_f {1 / sqrt(3), -1 / sqrt(3), 1 / sqrt(3)};
    vec3 point_g {1 / sqrt(3), 1 / sqrt(3), -1 / sqrt(3)};
    vec3 point_h {1 / sqrt(3), 1 / sqrt(3), 1 / sqrt(3)};

    // 6 sides means 12 polygons of triangles
    // Create with shifting
    polygons.push_back({point_a * r + center, point_b * r + center, point_d * r + center, color});
    polygons.push_back({point_a * r + center, point_c * r + center, point_d * r + center, color});
    polygons.push_back({point_b * r + center, point_f * r + center, point_h * r + center, color});
    polygons.push_back({point_b * r + center, point_d * r + center, point_h * r + center, color});
    polygons.push_back({point_e * r + center, point_f * r + center, point_h * r + center, color});
    polygons.push_back({point_e * r + center, point_g * r + center, point_h * r + center, color});
    polygons.push_back({point_a * r + center, point_e * r + center, point_g * r + center, color});
    polygons.push_back({point_a * r + center, point_c * r + center, point_g * r + center, color});
    polygons.push_back({point_a * r + center, point_b * r + center, point_f * r + center, color});
    polygons.push_back({point_a * r + center, point_e * r + center, point_f * r + center, color});
    polygons.push_back({point_c * r + center, point_d * r + center, point_h * r + center, color});
    polygons.push_back({point_c * r + center, point_g * r + center, point_h * r + center, color});
    // cout << "Creating hexahedron done\n";
}

void icosahedron(vec3 center, double radius, vec3 color, vector<polygon> &polygons) {
    double atctan_1_2 = 26.565; // arctan(1/2) ~ +-26.57
    double angle = M_PI * atctan_1_2 / 180;
    double segment_angle = M_PI * 72 / 180;
    double current_angle = 0.0;

    vector<vec3> vertices(12);
    vertices[0] = {0, radius, 0};
    vertices[11] = {0, -radius, 0};

    for (int i = 1; i < 6; i++) {
        vertices[i] = {radius * sin(current_angle) * cos(angle),
                       radius * sin(angle),
                       radius * cos(current_angle) * cos(angle)};
        current_angle += segment_angle;
    }

    current_angle = M_PI * 36 / 180;

    for (int i = 6; i < 11; i++) {
        vertices[i] = {radius * sin(current_angle) * cos(-angle),
                       radius * sin(-angle),
                       radius * cos(current_angle) * cos(-angle)};
        current_angle += segment_angle;
    }

    for (auto &j: vertices) {
        j.x = j.x + center.x;
        j.y = j.y + center.y;
        j.z = j.z + center.z;
    }

    polygons.push_back({vertices[10], vertices[6], vertices[1], color});
    polygons.push_back({vertices[9], vertices[10], vertices[5], color});
    polygons.push_back({vertices[8], vertices[9], vertices[4], color});
    polygons.push_back({vertices[7], vertices[8], vertices[3], color});
    polygons.push_back({vertices[6], vertices[7], vertices[2], color});

    polygons.push_back({vertices[1], vertices[5], vertices[10], color});
    polygons.push_back({vertices[5], vertices[4], vertices[9], color});
    polygons.push_back({vertices[4], vertices[3], vertices[8], color});
    polygons.push_back({vertices[3], vertices[2], vertices[7], color});
    polygons.push_back({vertices[2], vertices[1], vertices[6], color});

    polygons.push_back({vertices[11], vertices[6], vertices[10], color});
    polygons.push_back({vertices[11], vertices[10], vertices[9], color});
    polygons.push_back({vertices[11], vertices[9], vertices[8], color});
    polygons.push_back({vertices[11], vertices[8], vertices[7], color});
    polygons.push_back({vertices[11], vertices[7], vertices[6], color});

    polygons.push_back({vertices[0], vertices[5], vertices[1], color});
    polygons.push_back({vertices[0], vertices[4], vertices[5], color});
    polygons.push_back({vertices[0], vertices[3], vertices[4], color});
    polygons.push_back({vertices[0], vertices[2], vertices[3], color});
    polygons.push_back({vertices[0], vertices[1], vertices[2], color});
}

void dodecahedron(vec3 center, double r, vec3 color, vector<polygon> &polygons) { // +++++++
    // cout << "Creating dodecahedron\n";

    color = normalise_color(color);
    double a = (1 + sqrt(5)) / 2;
    double b = 2 / (1 + sqrt(5));
    // 20 vertices and 12 * 3 polygons (because pentagon == 3 triangles)
    vector<vec3> vertices {{-b, 0, a} ,
                                   { b, 0, a} ,
                                   {-1, 1, 1} ,
                                   { 1, 1, 1} ,
                                   { 1, -1, 1} ,
                                   {-1, -1, 1} ,
                                   { 0, -a, b} ,
                                   { 0, a, b} ,
                                   {-a, -b, 0} ,
                                   {-a, b, 0} ,
                                   { a, b, 0} ,
                                   { a, -b, 0} ,
                                   { 0, -a, -b} ,
                                   { 0, a, -b} ,
                                   { 1, 1, -1} ,
                                   { 1, -1, -1} ,
                                   {-1, -1, -1} ,
                                   {-1, 1, -1} ,
                                   { b, 0, -a} ,
                                   {-b, 0, -a}
    };

    for (auto &j: vertices) {
        j.x /= sqrt(3);
        j.y /= sqrt(3);
        j.z /= sqrt(3);
    }

    // Shifting
    for (auto &j: vertices) {
        j.x = j.x * r + center.x;
        j.y = j.y * r + center.y;
        j.z = j.z * r + center.z;
    }

    // Applying 36 polygons
    polygons.push_back({vertices[4], vertices[0], vertices[6], color});
    polygons.push_back({vertices[0], vertices[5], vertices[6], color});
    polygons.push_back({vertices[0], vertices[4], vertices[1], color});
    polygons.push_back({vertices[0], vertices[3], vertices[7], color});
    polygons.push_back({vertices[2], vertices[0], vertices[7], color});
    polygons.push_back({vertices[0], vertices[1], vertices[3], color});
    polygons.push_back({vertices[10], vertices[1], vertices[11], color});
    polygons.push_back({vertices[3], vertices[1], vertices[10], color});
    polygons.push_back({vertices[1], vertices[4], vertices[11], color});
    polygons.push_back({vertices[5], vertices[0], vertices[8], color});
    polygons.push_back({vertices[0], vertices[2], vertices[9], color});
    polygons.push_back({vertices[8], vertices[0], vertices[9], color});
    polygons.push_back({vertices[5], vertices[8], vertices[16], color});
    polygons.push_back({vertices[6], vertices[5], vertices[12], color});
    polygons.push_back({vertices[12], vertices[5], vertices[16], color});
    polygons.push_back({vertices[4], vertices[12], vertices[15], color});
    polygons.push_back({vertices[4], vertices[6], vertices[12], color});
    polygons.push_back({vertices[11], vertices[4], vertices[15], color});
    polygons.push_back({vertices[2], vertices[13], vertices[17], color});
    polygons.push_back({vertices[2], vertices[7], vertices[13], color});
    polygons.push_back({vertices[9], vertices[2], vertices[17], color});
    polygons.push_back({vertices[13], vertices[3], vertices[14], color});
    polygons.push_back({vertices[7], vertices[3], vertices[13], color});
    polygons.push_back({vertices[3], vertices[10], vertices[14], color});
    polygons.push_back({vertices[8], vertices[17], vertices[19], color});
    polygons.push_back({vertices[16], vertices[8], vertices[19], color});
    polygons.push_back({vertices[8], vertices[9], vertices[17], color});
    polygons.push_back({vertices[14], vertices[11], vertices[18], color});
    polygons.push_back({vertices[11], vertices[15], vertices[18], color});
    polygons.push_back({vertices[10], vertices[11], vertices[14], color});
    polygons.push_back({vertices[12], vertices[19], vertices[18], color});
    polygons.push_back({vertices[15], vertices[12], vertices[18], color});
    polygons.push_back({vertices[12], vertices[16], vertices[19], color});
    polygons.push_back({vertices[19], vertices[13], vertices[18], color});
    polygons.push_back({vertices[17], vertices[13], vertices[19], color});
    polygons.push_back({vertices[13], vertices[14], vertices[18], color});

    // cout << "Creating dodecahedron done\n";
}

void scene(vec3 a, vec3 b, vec3 c, vec3 d, vec3 color,
           vector<polygon> &polygons) {
    // cout << "Creating scene\n";
    color = normalise_color(color);
    polygons.push_back(polygon{a, b, c, color});
    polygons.push_back(polygon{c, d, a, color});
}

int cpu_mode(vec3 p_c, vec3 p_v, int w, int ssaa_w, int h, int ssaa_h, double fov, uchar4* pixels,
             uchar4* pixels_ssaa, vec3 light_pos, vec3 light_col, polygon* polygons, int n, int ssaa_multiplier) {
    render_cpu(p_c, p_v, ssaa_w, ssaa_h, fov, pixels_ssaa, light_pos, light_col, polygons, n);
    ssaa_cpu(pixels, w, h, ssaa_multiplier, pixels_ssaa);

    return 0;
}

int gpu_mode(vec3 p_c, vec3 p_v, int w, int ssaa_w, int h, int ssaa_h, double fov, uchar4* pixels,
             uchar4* pixels_ssaa, vec3 light_pos, vec3 light_col, polygon* polygons, int n, int ssaa_multiplier) {
//    cerr << "Allocate pixels\n";
    // Allocating on gpu
    uchar4* gpu_pixels;
    CUDA_ERROR(hipMalloc((uchar4**)(&gpu_pixels), w * h * sizeof(uchar4)));
    CUDA_ERROR(hipMemcpy(gpu_pixels, pixels, w * h * sizeof(uchar4), hipMemcpyHostToDevice));
//    cerr << "Allocate ssaa pixels\n";
    uchar4* gpu_pixels_ssaa;
    CUDA_ERROR(hipMalloc((uchar4**)(&gpu_pixels_ssaa), ssaa_w * ssaa_h * sizeof(uchar4)));
    CUDA_ERROR(hipMemcpy(gpu_pixels_ssaa, pixels_ssaa, ssaa_w * ssaa_h * sizeof(uchar4), hipMemcpyHostToDevice));
//    cerr << "Allocate polygons\n";
    polygon* gpu_polygons;
    CUDA_ERROR(hipMalloc((polygon**)(&gpu_polygons), n * sizeof(polygon)));
    CUDA_ERROR(hipMemcpy(gpu_polygons, polygons, n * sizeof(polygon), hipMemcpyHostToDevice));
//    cerr << "Start render\n";
    // Rendering
    render_gpu <<< 128, 128 >>> (p_c, p_v, ssaa_w, ssaa_h, fov, gpu_pixels_ssaa, light_pos, light_col, gpu_polygons, n);
    hipDeviceSynchronize();
    CUDA_ERROR(hipGetLastError());
//    cerr << "Start ssaa\n";
    // Ssaa smoothing algo
    ssaa_gpu <<< 128, 128 >>> (gpu_pixels, w, h, ssaa_multiplier, gpu_pixels_ssaa);
    hipDeviceSynchronize();
    CUDA_ERROR(hipGetLastError());
    CUDA_ERROR(hipMemcpy(pixels, gpu_pixels, w * h * sizeof(uchar4), hipMemcpyDeviceToHost));

    // Free memory
    CUDA_ERROR(hipFree(gpu_pixels));
    CUDA_ERROR(hipFree(gpu_pixels_ssaa));
    CUDA_ERROR(hipFree(gpu_polygons));

    return 0;
}

int main(int argc, char* argv[]) {
    string mode;
    if (argv[1])
        mode = argv[1];
    bool is_gpu = true;

    if (argc > 2) {
        cout << "Incorrect params. Please use '--help' for help\n";
        return 0;
    }

    if (argc == 1 || mode == "--gpu")
        is_gpu = true;

    if (mode == "--cpu")
        is_gpu = false;

    if (mode == "--default") {
        cout << "10\n"
                "./frames_data\n"
                "640 480 120\n"
                "7.0 3.0 0.0 2.0 1.0 2.0 6.0 1.0 0.0 0.0\n"
                "2.0 0.0 0.0 0.5 0.1 1.0 4.0 1.0 0.0 0.0\n"
                "4.0 4.0 0.0 0.5 0.0 1.0 2.0 0.0 0.0 0.0\n"
                "1.0 1.0 0.0 0.5 0.0 1.0 2.0 0.0 0.0 0.0\n"
                "-2.5 -2.5 0.0 0.5 1.0 1.0 2.0 0.0 0.0 0.0\n"
                "-10.0 -10.0 -1.0 -10.0 10.0 -1.0 10.0 10.0 -1.0 10.0 -10.0 -1.0 temp 0.0 0.9 0.0 0.5\n"
                "1\n"
                "100 100 100 1.0 1.0 1.0\n"
                "1 3\n";
        return 0;
    }

    if (mode == "--help") {
        cout << "<---------------   HELP   --------------->\n"
                "Start program without args will cause computation in gpu mode\n"
                "--cpu     For computation with using cpu\n"
                "--gpu     For computation with using gpu\n"
                "--default Print best configuration for input data\n"
                "--help    For help\n"
                "<---------------END OF HELP--------------->\n";
        return 0;
    }

    int total_frames, width, height, fov;
    string path_to_frames;

    double r_0c, z_0c, phi_0c;
    double A_rc, A_zc;
    double w_rc, w_zc, w_phic;
    double p_rc, p_zc;

    double r_0v, z_0v, phi_0v;
    double A_rv, A_zv;
    double w_rv, w_zv, w_phiv;
    double p_rv, p_zv;

    vec3 center, color;
    double radius;

    string unused;

    vec3 scene_a, scene_b, scene_c, scene_d;
    vec3 light_pos, light_col;

    vector<polygon> polygons;
    polygon *polygons_as_array;
    uchar4 *pixels = nullptr;
    uchar4 *pixels_ssaa = nullptr;


    int n_lights; // Should be 1 (1 light)
    int recursion_step; // Should be 1 (unused)
    int ssaa_multiplier;

    // Frames
    cin >> total_frames;
    cin >> path_to_frames;
    cin >> width >> height >> fov;

    // Camera trajectory
    cin >> r_0c >> z_0c >> phi_0c;
    cin >> A_rc >> A_zc;
    cin >> w_rc >> w_zc >> w_phic;
    cin >> p_rc >> p_zc;

//    cerr << r_0c << " " << z_0c << " " << phi_0c << "\n";
//    cerr << A_rc << " " << A_zc << "\n";
//    cerr << w_rc << " " << w_zc<< " "  << w_phic << "\n";
//    cerr << p_rc << " " << p_zc << "\n";

    cin >> r_0v >> z_0v >> phi_0v;
    cin >> A_rv >> A_zv;
    cin >> w_rv >> w_zv >> w_phiv;
    cin >> p_rv >> p_zv;

//    cerr << r_0v << " " << z_0v << " " << phi_0v << "\n";
//    cerr << A_rv << " " << A_zv << "\n";
//    cerr << w_rv << " " << w_zv << " " << w_phiv << "\n";
//    cerr << p_rv << " " << p_zv << "\n";

    // Figures params with creating
    cin >> center.x >> center.y >> center.z >> color.x >> color.y >> color.z >> radius >> unused >> unused >> unused;
    hexahedron(center, radius, color, polygons);
    cin >> center.x >> center.y >> center.z >> color.x >> color.y >> color.z >> radius >> unused >> unused >> unused;
    icosahedron(center, radius, color, polygons);
    cin >> center.x >> center.y >> center.z >> color.x >> color.y >> color.z >> radius >> unused >> unused >> unused;
    dodecahedron(center, radius, color, polygons);

    // Scene
    cin >> scene_a.x >> scene_a.y >> scene_a.z;
    cin >> scene_b.x >> scene_b.y >> scene_b.z;
    cin >> scene_c.x >> scene_c.y >> scene_c.z;
    cin >> scene_d.x >> scene_d.y >> scene_d.z;

    cin >> unused;
    cin >> color.x >> color.y >> color.z;
    cin >> unused;
    scene(scene_a, scene_b, scene_c, scene_d, color, polygons);

    // Lights
    cin >> n_lights;
    cin >> light_pos.x >> light_pos.y >> light_pos.z;
    cin >> light_col.x >> light_col.y >> light_col.z;

    // Recursion
    cin >> recursion_step;
//    cerr << recursion_step << "\n";

    // SSAA params
    cin >> ssaa_multiplier;
//    cerr << ssaa_multiplier << "\n";

    int ssaa_width = width * ssaa_multiplier;
    int ssaa_height = height * ssaa_multiplier;

    pixels = new uchar4[ssaa_width * ssaa_height];
    pixels_ssaa = new uchar4[ssaa_width * ssaa_height]; // cpu

    polygons_as_array = polygons.data();
    int total_polygons = polygons.size();

//    for (int i = 0; i < polygons.size(); ++i) {
//        cerr << "p1:" << polygons[i].p1.x << " " << polygons[i].p1.y << " " << polygons[i].p1.z << "\n";
//        cerr << "p2:" << polygons[i].p2.x << " " << polygons[i].p2.y << " " << polygons[i].p2.z << "\n";
//        cerr << "p3:" << polygons[i].p3.x << " " << polygons[i].p3.y << " " << polygons[i].p3.z << "\n";
//        cerr << "color:" << polygons[i].color.x << " " << polygons[i].color.y << " " << polygons[i].color.z << "\n";
//    }

    cout << "Start rendering. Total polygons: " << total_polygons << ". Frame size: " << width << "x" << height;
    cout << ". Total frames: " << total_frames << "\n";
    cout << "|\tIteration number\t|\t time in ms\t|\ttotal rays |\n";

    double r_c, z_c, phi_c , r_v, z_v, phi_v;
    vec3 p_c, p_v;
    int sum_of_rays;

    double total_duration_time = 0;
    for (int i = 0; i < total_frames; i++) {
        auto start = chrono::steady_clock::now();
        double time_step = 2.0 * M_PI / total_frames;
        double cur_time = i * time_step;

        // Movement
        r_c = r_0c + A_rc * sin(w_rc * cur_time + p_rc);
        z_c = z_0c + A_zc * sin(w_zc * cur_time + p_zc);
        phi_c = phi_0c + w_phic * cur_time;

        r_v = r_0v + A_rv * sin(w_rv * cur_time + p_rv);
        z_v = z_0v + A_zv * sin(w_zv * cur_time + p_zv);
        phi_v = phi_0v + w_phiv * cur_time;

        p_c = { r_c * cos(phi_c), r_c * sin(phi_c), z_c };
        p_v = { r_v * cos(phi_v), r_v * sin(phi_v), z_v };

        // Total sum of rays (will be the same coz of recursion)
        sum_of_rays = ssaa_width * ssaa_height;

        int res;
        if (is_gpu)
            res = gpu_mode(p_c, p_v, width, ssaa_width, height, ssaa_height, (double)fov, pixels, pixels_ssaa,
                           light_pos, light_col, polygons_as_array, total_polygons, ssaa_multiplier);
        else
            res = cpu_mode(p_c, p_v, width, ssaa_width, height, ssaa_height, (double)fov, pixels, pixels_ssaa,
                           light_pos, light_col, polygons_as_array, total_polygons, ssaa_multiplier);
        if (res)
            cout << "An error occurred. Check output\n";

        auto end = chrono::steady_clock::now();
        cout << "|\tIteration " << i + 1 << " of " << total_frames << "\t|\t";
        double iteration_time = ((double)chrono::duration_cast<chrono::microseconds>(end - start).count()) / 1000.0;
        total_duration_time += iteration_time;
        cout << iteration_time << "ms\t|\t";
        cout << sum_of_rays << "\t\t|\n";

        string frame_name = path_to_frames + "/" + to_string(i) + ".data";
        FILE* f = fopen(frame_name.c_str(), "wb");
//        fwrite(&ssaa_width, sizeof(int), 1, f);
//        fwrite(&ssaa_height, sizeof(int), 1, f);
//        fwrite(pixels_ssaa, sizeof(uchar4), ssaa_width * ssaa_height, f);
        fwrite(&width, sizeof(int), 1, f); // тут падает
        fwrite(&height, sizeof(int), 1, f);
        fwrite(pixels, sizeof(uchar4), width * height, f);
        fclose(f);
    }

    if (pixels)
        delete[] pixels;
    if (pixels_ssaa)
        delete[] pixels_ssaa;

    cout << "Done with total duration: " << total_duration_time << "ms\n";
    return 0;
}